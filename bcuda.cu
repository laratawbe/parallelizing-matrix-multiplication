
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>



// __global__ void matrixMultiplication(float *A, float *B, float *C, int m, int n, int k) {
//     int row = blockIdx.y * blockDim.y + threadIdx.y;
//     int col = blockIdx.x * blockDim.x + threadIdx.x;

//     if (row < m && col < k) {
//         float sum = 0.0;
//         for (int i = 0; i < n; i++) {
//             sum += A[row * n + i] * B[i * k + col];
//         }
//         C[row * k + col] = sum;
//     }
// }
__global__ void matrixMultiplication(float *a, float *b, float *c, int n);

int main(){
    int n = 1000;
    float *a, *b, *c;
    int size = n * n * sizeof(float);
    a = (float *)malloc(size);
    b = (float *)malloc(size);
    
    c = (float *)malloc(size);
    srand(time(NULL));
    for (int i = 0; i < n * n; i++){
        a[i] = (float)rand() / RAND_MAX;
        b[i] = (float)rand() / RAND_MAX;
    }

    float *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);

    hipMalloc((void **)&d_c, size);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 dimGrid((n + 15) / 16, (n + 15) / 16);
    dim3 dimBlock(16, 16);
    clock_t start_time = clock();

    matrixMultiplication<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);
    hipDeviceSynchronize();

    clock_t end_time = clock();
    double elapsed_time = ((double)(end_time - start_time)) / CLOCKS_PER_SEC;
    printf("time: %.7f seconds\n", elapsed_time);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}

__global__ void matrixMultiplication(float *a, float *b, float *c, int n){
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row  < n && col  < n){
        float sum = 0.0;

        for (int i = 0; i < n; i++){
            sum += a[row*n + i] * b[i*n + col];
        }
        c[row*n + col] = sum;
    }
}
