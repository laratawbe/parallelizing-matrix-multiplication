
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define MATRIX_SIZE 1000
#define TILE_SIZE 16

__global__ void matrixMultiplication(float *matrixA, float *matrixB, float *matrixC, int n);


int main(){
    float *a, *b, *c;
    int n = MATRIX_SIZE;
    int size = n * n * sizeof(float);

    a = (float *)malloc(size);
    b = (float *)malloc(size);
    c = (float *)malloc(size);

    srand(time(NULL));
    for (int i = 0; i < n * n; ++i){
        a[i] = (float)rand() / RAND_MAX;
        b[i] = (float)rand() / RAND_MAX;
    }

    float *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size);

    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    dim3 dimGrid((n + TILE_SIZE - 1) / TILE_SIZE, (n + TILE_SIZE - 1) / TILE_SIZE);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    clock_t start_time = clock();

    matrixMultiplication<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);
    hipDeviceSynchronize();

    clock_t end_time = clock();
    double elapsed_time = ((double)(end_time - start_time)) / CLOCKS_PER_SEC;
    printf("time: %.7f seconds\n", elapsed_time);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}

__global__ void matrixMultiplication(float *matrixA, float *matrixB, float *matrixC, int n){
    float partialSum = 0.00;
    __shared__ float sharedMatrixA[TILE_SIZE][TILE_SIZE];

    __shared__ float sharedMatrixB[TILE_SIZE][TILE_SIZE];
    for (int tile = 0; tile < (n + TILE_SIZE - 1) / TILE_SIZE; tile++){
    
        int tileRow = blockIdx.y * TILE_SIZE + threadIdx.y;
        int tileCol = tile * TILE_SIZE + threadIdx.x;

        if (tileRow < n && tileCol < n){
            sharedMatrixA[threadIdx.y][threadIdx.x] = matrixA[tileRow * n + tileCol];
        }
        else{
            sharedMatrixA[threadIdx.y][threadIdx.x] = 0.0;
        }
        tileRow = tile * TILE_SIZE + threadIdx.y;
        tileCol = blockIdx.x * TILE_SIZE + threadIdx.x;

        if (tileRow < n && tileCol < n){
            sharedMatrixB[threadIdx.y][threadIdx.x] = matrixB[tileRow * n + tileCol]; //review
        }
            
        else{
            sharedMatrixB[threadIdx.y][threadIdx.x] = 0.0;

        }
            
        __syncthreads();

        for (int k = 0; k < TILE_SIZE; k++){
            partialSum += sharedMatrixA[threadIdx.y][k] * sharedMatrixB[k][threadIdx.x];
        }
        __syncthreads();
    }

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;

    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    if (row < n && col < n){
        matrixC[row*n + col] = partialSum;
    }
}

